#include "hip/hip_runtime.h"
#include <THC/THCBlas.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDABlas.h>

#include <algorithm>
#include <mutex>

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif

/* Level 2 */

void adjustLdLevel2(int64_t m, int64_t n, int64_t *lda)
{
  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  // TODO: why does Level3 check trans but this doesn't?
  if (n <= 1)
    *lda = std::max<int64_t>(m, 1);
}

void THCudaBlas_Sger(THCState *state, int64_t m, int64_t n, float alpha, float *x, int64_t incx, float *y, int64_t incy, float *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, int64_t m, int64_t n, double alpha, double *x, int64_t incx, double *y, int64_t incy, double *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

// Check https://github.com/pytorch/pytorch/issues/22078
// for information about the bug. We don't know the exact conditions that trigger it,
// but using Sgemm or Hgemm on Maxwell or Pascal seems to be a
// necessary condition.
static void checkCuda90Bug(int i_m, int i_n, int i_k)
{
#if CUDA_VERSION < 9200 && CUDA_VERSION >= 9000
  static std::once_flag alreadyWarned;
  const int LIMIT = 1 << 21;
  if (i_m > LIMIT || i_n > LIMIT || i_k > LIMIT) {
    hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
    if (prop->major == 5 || prop->major == 6) {
      std::call_once(alreadyWarned, []() {
        TORCH_WARN("Matrix multiplication for dimensions larger than 2^21 has known bugs on your combination of CUDA version and device type. Please consider upgrading to CUDA 9.2 or later.");
      });
    }
  }
#endif
}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  checkCuda90Bug((int)m, (int)n, (int)k);
  at::cuda::blas::gemm<float>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::Half alpha, at::Half *a, int64_t lda, at::Half *b, int64_t ldb, at::Half beta, at::Half *c, int64_t ldc)
{
  checkCuda90Bug((int)m, (int)n, (int)k);
  at::cuda::blas::gemm<at::Half>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

#ifdef __HIP_PLATFORM_HCC__
void THCudaBlas_Bgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::BFloat16 alpha, at::BFloat16 *a, int64_t lda, at::BFloat16 *b, int64_t ldb, at::BFloat16 beta, at::BFloat16 *c, int64_t ldc)
{
  at::cuda::blas::gemm<at::BFloat16>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}
#endif

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  at::cuda::blas::gemm<double>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}
